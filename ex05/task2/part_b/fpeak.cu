// File       : fpeak.cu
// Created    : Sat May 01 2021 09:57:06 AM (+0200)
// Description: Floating point peak performance kernel
// Copyright 2021 ETH Zurich. All Rights Reserved.

#include "CUDATimer.cuh"
#include <stdio.h>

typedef float Real;

// TODO: Define the nominal floating point performance of your target GPU in
// gigaflop
#define PMAX (1)

// TODO: Define a GPU kernel to attain peak floating point performance

int main(void)
{
    GPUtimer timer;
    timer.start();

    // TODO: Invoke your kernel here

    timer.stop();

    hipDeviceSynchronize();

    // TODO: Compute the number of gigaflop your kernel has performed within
    // the measured time
    const double Gflop = 1;

    const double kTime = timer.elapsed() * 1.0e-3; // seconds
    const double Perf = Gflop / kTime; // Gflops
    const double frac = Perf / PMAX;   // fraction of peak
    printf("Performance: %f Gflops (%4.1f%% of Peak)\n", Perf, frac * 100);
    printf("Kernel Time: %f s\n", kTime);

    return 0;
}
