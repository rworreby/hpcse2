#include <hip/hip_runtime.h>
#include <cstdio>

__global__ void computeForcesKernel(int N, const double3 *p, double3 *f) {

    for(int idx = blockIdx.x * blockDim.x + threadIdx.x;
            idx < N;
            idx += gridDim.x * blockDim.x){

        f[idx] = double3{0.0, 0.0, 0.0};
        for (int i = 0; i < N; ++i) {
            if(i != idx){
                double dx = p[i].x - p[idx].x;
                double dy = p[i].y - p[idx].y;
                double dz = p[i].z - p[idx].z;
                double r = sqrt(dx * dx + dy * dy + dz * dz);
                double inv_r = 1.0 / r;
                f[idx].x += dx * inv_r * inv_r * inv_r;
                f[idx].y += dy * inv_r * inv_r * inv_r;
                f[idx].z += dz * inv_r * inv_r * inv_r;
            }
        }
    }
}

void computeForces(int N, const double3 *p, double3 *f) {
    constexpr int numThreads = 1024;
    int numBlocks = (N + numThreads - 1) / numThreads;
    computeForcesKernel<<<numBlocks, numThreads>>>(N, p, f);
}
