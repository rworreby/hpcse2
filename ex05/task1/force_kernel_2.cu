#include <hip/hip_runtime.h>
#include <cstdio>

// TODO Improve this kernel.
//      Create a new file for each new category of optimizations you
//      successively apply {force_kernel_2.cu, force_kernel_3.cu, ...}
__global__ void computeForcesKernel(int N, const double3 *p, double3 *f) {

    for(int idx = blockIdx.x * blockDim.x + threadIdx.x;
            idx < N;
            idx += gridDim.x * blockDim.x){

        double f_temp_x = 0.0;
        double f_temp_y = 0.0;
        double f_temp_z = 0.0;
        for (int i = 0; i < N; ++i) {
            if(i != idx){
                double dx = p[i].x - p[idx].x;
                double dy = p[i].y - p[idx].y;
                double dz = p[i].z - p[idx].z;
                double r = sqrt(dx * dx + dy * dy + dz * dz);
                double inv_r = 1.0 / r;
                double inv_r_3 = inv_r * inv_r * inv_r;

                f_temp_x += dx * inv_r_3;
                f_temp_y += dy * inv_r_3;
                f_temp_z += dz * inv_r_3;
            }
        }
        f[idx].x = f_temp_x;
        f[idx].y = f_temp_y;
        f[idx].z = f_temp_z;
    }
}

void computeForces(int N, const double3 *p, double3 *f) {
    constexpr int numThreads = 1024;
    int numBlocks = (N + numThreads - 1) / numThreads;
    computeForcesKernel<<<numBlocks, numThreads>>>(N, p, f);
}
